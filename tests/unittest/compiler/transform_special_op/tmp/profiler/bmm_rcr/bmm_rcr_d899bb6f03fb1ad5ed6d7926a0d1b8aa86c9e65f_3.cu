#include "hip/hip_runtime.h"

size_t GLOBAL_WORKSPACE_SIZE = 0;

#include <sstream>


#include <iostream>
#include <memory>
#include <random>
#include <vector>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm_universal.h"
#include "cutlass/gemm/kernel/gemm_grouped.h"
#include "cutlass/gemm/kernel/default_gemm_grouped.h"
#include "cutlass/gemm/device/gemm_grouped.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "cutlass/util/device_memory.h"

#include "cutlass/gemm/gemm.h"
#include "cutlass/numeric_types.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/epilogue/collective/collective_builder.hpp"

using bfloat16 = hip_bfloat16;



#define CUTLASS_CHECK(status)                                                         \
  {                                                                                   \
    cutlass::Status error = status;                                                   \
    if (error != cutlass::Status::kSuccess) {                                         \
      auto msg = std::string("[") + __FILE__ + "] Got cutlass error: " +              \
          cutlassGetStatusString(error) + " at: " + std::to_string(__LINE__);         \
      std::cerr << msg << std::endl;                                                  \
      throw std::runtime_error(msg);                                                  \
    }                                                                                 \
  }



  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_256x128_32x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_256x128_32x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_0 = Operation_cutlass_tensorop_f16_s16816gemm_f16_256x128_32x3_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x256_32x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_128x256_32x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 256, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_1 = Operation_cutlass_tensorop_f16_s16816gemm_f16_128x256_32x3_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_256x64_32x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_256x64_32x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 64, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_2 = Operation_cutlass_tensorop_f16_s16816gemm_f16_256x64_32x3_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_256x64_32x4_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_256x64_32x4_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 64, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    4,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_3 = Operation_cutlass_tensorop_f16_s16816gemm_f16_256x64_32x4_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_64x256_32x4_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_64x256_32x4_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 256, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    4,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_4 = Operation_cutlass_tensorop_f16_s16816gemm_f16_64x256_32x4_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x128_32x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_128x128_32x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_5 = Operation_cutlass_tensorop_f16_s16816gemm_f16_128x128_32x3_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x128_32x4_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_128x128_32x4_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    4,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_6 = Operation_cutlass_tensorop_f16_s16816gemm_f16_128x128_32x4_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x128_32x5_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_128x128_32x5_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 32>,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    5,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_7 = Operation_cutlass_tensorop_f16_s16816gemm_f16_128x128_32x5_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x64_32x6_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_128x64_32x6_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 64, 32>,
    cutlass::gemm::GemmShape<64, 32, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    6,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_8 = Operation_cutlass_tensorop_f16_s16816gemm_f16_128x64_32x6_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_64x128_32x6_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_64x128_32x6_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 128, 32>,
    cutlass::gemm::GemmShape<32, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    6,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_9 = Operation_cutlass_tensorop_f16_s16816gemm_f16_64x128_32x6_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_64x64_32x10_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_64x64_32x10_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 32>,
    cutlass::gemm::GemmShape<32, 32, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    10,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_10 = Operation_cutlass_tensorop_f16_s16816gemm_f16_64x64_32x10_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_256x128_64x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_256x128_64x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 128, 64>,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_11 = Operation_cutlass_tensorop_f16_s16816gemm_f16_256x128_64x3_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x256_64x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_128x256_64x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 256, 64>,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_12 = Operation_cutlass_tensorop_f16_s16816gemm_f16_128x256_64x3_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_256x64_64x4_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_256x64_64x4_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 64, 64>,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    4,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_13 = Operation_cutlass_tensorop_f16_s16816gemm_f16_256x64_64x4_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_64x256_64x4_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_64x256_64x4_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 256, 64>,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    4,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_14 = Operation_cutlass_tensorop_f16_s16816gemm_f16_64x256_64x4_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x128_64x4_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_128x128_64x4_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 64>,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    4,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_15 = Operation_cutlass_tensorop_f16_s16816gemm_f16_128x128_64x4_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_256x64_64x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_256x64_64x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 64, 64>,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_16 = Operation_cutlass_tensorop_f16_s16816gemm_f16_256x64_64x3_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_64x256_64x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_64x256_64x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 256, 64>,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_17 = Operation_cutlass_tensorop_f16_s16816gemm_f16_64x256_64x3_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x128_64x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_128x128_64x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 128, 64>,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_18 = Operation_cutlass_tensorop_f16_s16816gemm_f16_128x128_64x3_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_128x64_64x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_128x64_64x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 64, 64>,
    cutlass::gemm::GemmShape<64, 32, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_19 = Operation_cutlass_tensorop_f16_s16816gemm_f16_128x64_64x3_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_64x128_64x3_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_64x128_64x3_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 128, 64>,
    cutlass::gemm::GemmShape<32, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    3,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_20 = Operation_cutlass_tensorop_f16_s16816gemm_f16_64x128_64x3_tn_align8;


  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_64x64_64x5_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_64x64_64x5_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<32, 32, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    5,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using GemmInstance_21 = Operation_cutlass_tensorop_f16_s16816gemm_f16_64x64_64x5_tn_align8;

template <typename GemmInstance>
void bmm (
    GemmInstance& gemm_op,
    void* a_ptr,
    void* b_ptr,
    void* c_ptr,
    uint8_t* workspace,
    int64_t* a_dim0,
    int64_t* a_dim1,
    int64_t* a_dim2,
    int64_t* b_dim0,
    int64_t* b_dim1,
    int64_t* b_dim2,
    int64_t* c_dim0,
    int64_t* c_dim1,
    int64_t* c_dim2,
    hipStream_t stream
  ) {
  
  int64_t B = (*a_dim0);

  int64_t M = (*a_dim1);

  int64_t N = (*b_dim1);

  int64_t K = (*a_dim2);
  
  
  
  
  int64_t a_size = 1;

    a_size *= *a_dim0;

    a_size *= *a_dim1;

    a_size *= *a_dim2;

  if (a_size != 0 && !a_ptr) {
    throw std::runtime_error("input a is null!");
  }

  int64_t b_size = 1;

    b_size *= *b_dim0;

    b_size *= *b_dim1;

    b_size *= *b_dim2;

  if (b_size != 0 && !b_ptr) {
    throw std::runtime_error("input b is null!");
  }

  int64_t c_size = 1;

    c_size *= *c_dim0;

    c_size *= *c_dim1;

    c_size *= *c_dim2;

  if (c_size != 0) {
    if (!c_ptr) {
      throw std::runtime_error("input c is null!");
    }
  } else {
    // output is empty and safe to return
    return;
  }

  // One of the input tensor are empty
  if (a_size == 0 || b_size == 0) {
    return;
  }

  
//  TODO: cast to right dtype
  using ElementComputeEpilogue = typename GemmInstance::ElementAccumulator;

  using coord_t = cutlass::gemm::GemmCoord::Index;
  typename GemmInstance::Arguments arguments;

  if constexpr (cutlass::gemm::detail::IsCutlass3GemmKernel<typename GemmInstance::GemmKernel>::value) {
  arguments = {

    cutlass::gemm::GemmUniversalMode::kBatched,                                 // GemmUniversalMode mode
    {
        static_cast<coord_t>(M),
        static_cast<coord_t>(N),
        static_cast<coord_t>(K),
        static_cast<coord_t>(B)
    },                                                                          // ProblemShape problem_shape
    (cutlass::half_t*)(a_ptr),                                                          // ElementA const* ptr_A

    { K, cute::Int<1>{}, M * K },            // StrideA dA

    (cutlass::half_t*)(b_ptr),                                                          // ElementB const* ptr_B

    { K, cute::Int<1>{}, N * K },            // StrideB dB

    {
        {
            ElementComputeEpilogue(1.0),
            ElementComputeEpilogue(0)
        },                                                                      // typename ThreadEpilogueOp::Params thread
        (cutlass::half_t*)(c_ptr),                                                   // ElementC const* ptr_C

        { N, cute::Int<1>{}, M * N },  // StrideC dC

        (cutlass::half_t*)(c_ptr),                                                      // ElementD const* ptr_D

        { N, cute::Int<1>{}, M * N },        // StrideD dD

    },                                                                          // EpilogueArguments epilogue
  };
  } else {
  arguments = {

    cutlass::gemm::GemmUniversalMode::kBatched,                                                         // GemmUniversalMode mode
    {
        static_cast<coord_t>(M),
        static_cast<coord_t>(N),
        static_cast<coord_t>(K)
    },                                                                                                  // GemmCoord problem_size
    B,                                                                             // int batch_count
    {ElementComputeEpilogue(1.0), ElementComputeEpilogue(0)},  // typename EpilogueOutputOp::Params epilogue
    a_ptr,                                                                                  // void const * ptr_A
    b_ptr,                                                                                  // void const * ptr_B
    c_ptr,                                                                               // void const * ptr_C
    c_ptr,                                                                                  // void * ptr_D
    M * K,                                                                         // int64_t batch_stride_A
    N * K,                                                                         // int64_t batch_stride_B
    M * N,                                                                      // int64_t batch_stride_C
    M * N,                                                                         // int64_t batch_stride_D
    K,                                                                                    // typename LayoutA::Stride::LongIndex lda
    K,                                                                                    // typename LayoutB::Stride::LongIndex ldb
    N,                                                                                 // typename LayoutC::Stride::LongIndex ldc
    N,                                                                                    // typename LayoutC::Stride::LongIndex ldd
  };
  }


  size_t workspace_size = gemm_op.get_workspace_size(arguments);
  cutlass::device_memory::allocation<uint8_t> local_workspace(workspace_size);
  workspace = local_workspace.get();
  GLOBAL_WORKSPACE_SIZE = workspace_size;

  auto status = gemm_op.can_implement(arguments);
  CUTLASS_CHECK(status);
  status = gemm_op.initialize(arguments, workspace, stream);
  CUTLASS_CHECK(status);
  status = gemm_op(stream);
  CUTLASS_CHECK(status);
  return;
      std::cout << "input_ndims0: " << *a_dim0 << std::endl;
      std::cout << "input_ndims1: " << *a_dim1 << std::endl;
      std::cout << "input_ndims2: " << *a_dim2 << std::endl;
      std::cout << "weight_ndims0: " << *b_dim0 << std::endl;
      std::cout << "weight_ndims1: " << *b_dim1 << std::endl;
      std::cout << "weight_ndims2: " << *b_dim2 << std::endl;
      std::cout << "output_ndims0: " << *c_dim0 << std::endl;
      std::cout << "output_ndims1: " << *c_dim1 << std::endl;
      std::cout << "output_ndims2: " << *c_dim2 << std::endl;
  throw std::runtime_error(
      "Unsupported workload for this bmm specialization."
  );
}

template <typename DType>
struct ProfilerMemoryPool;

template <typename GemmInstance>
int benchmark_bmm (


    GemmInstance &gemm_op,
    const char *gemm_op_name,
    ProfilerMemoryPool<half>* memory_pool,
    uint8_t* global_workspace_,


    int64_t* a_dim0,

    int64_t* a_dim1,

    int64_t* a_dim2,


    int64_t* b_dim0,

    int64_t* b_dim1,

    int64_t* b_dim2,


    int64_t* c_dim0,

    int64_t* c_dim1,

    int64_t* c_dim2,

    hipStream_t stream

  ) {
  // warmup
  for (int i = 0; i < 5; ++i) {
    
{

bmm(

    gemm_op,

    memory_pool->RequestTensorByIdx(0),
    memory_pool->RequestTensorByIdx(1),


    memory_pool->RequestTensorByIdx(2),
    global_workspace_,

    a_dim0,

    a_dim1,

    a_dim2,


    b_dim0,

    b_dim1,

    b_dim2,


    c_dim0,

    c_dim1,

    c_dim2,

    stream
);
}
  }
  hipEvent_t events[2];
  for (auto & event : events) {
    hipEventCreate(&event);
  }
  hipEventRecord(events[0], stream);
  for (int i = 0; i < 10; ++i) {
    
{

bmm(

    gemm_op,

    memory_pool->RequestTensorByIdx(0),
    memory_pool->RequestTensorByIdx(1),


    memory_pool->RequestTensorByIdx(2),
    global_workspace_,

    a_dim0,

    a_dim1,

    a_dim2,


    b_dim0,

    b_dim1,

    b_dim2,


    c_dim0,

    c_dim1,

    c_dim2,

    stream
);
}
  }
  hipEventRecord(events[1], stream);
  hipEventSynchronize(events[1]);
  float runtime_ms = 0;
  hipEventElapsedTime(&runtime_ms, events[0], events[1]);
  for (auto event : events) {
    (void)hipEventDestroy(event);
  }
  // TODO: output workspace
  if (runtime_ms < 0.00001) {
      throw std::runtime_error(
      "OOB in cutlass."
    );
  }
  std::cout << "OP:" << gemm_op_name << ",";
  std::cout << "TIME:" << runtime_ms << ",";
  std::cout << "WS:" << GLOBAL_WORKSPACE_SIZE << std::endl;
  return 0;
}

template <typename DType>
struct ProfilerMemoryPool {
  ProfilerMemoryPool() : shared_input_tensor(false) {
    std::random_device rd;
    gen = std::mt19937(rd());
    uniform_dist = std::uniform_int_distribution<int64_t>(1, 48964896);
    offsets.reserve(512);
    strides.reserve(512);
    copies.reserve(512);
    ptrs.reserve(512);
    blobs.reserve(512);
  }
  ~ProfilerMemoryPool() {}

  int64_t ComputeMemPoolSize(size_t one_copy_sz, size_t ptr_max_sz, size_t l2_cache_bytes) {
    int times_covers_l2_cache = (int)std::ceil(l2_cache_bytes / sizeof(DType) / ptr_max_sz);
    int64_t mem_pool_sz = std::max(2, std::min(512, times_covers_l2_cache));
    size_t free_global_mem = 0;
    size_t total_global_mem = 0;
    hipError_t cuda_error = hipMemGetInfo(&free_global_mem, &total_global_mem);
    if (cuda_error != hipSuccess) {
      auto error_msg = std::string("Failed to invoke hipMemGetInfo: ") +
          hipGetErrorName(cuda_error) + ", at " + __FILE__;
      throw std::runtime_error(error_msg);
    }
    size_t single_copy_nbytes = one_copy_sz * sizeof(DType);
    while (mem_pool_sz > 0) {
      size_t nbytes = single_copy_nbytes * mem_pool_sz;
      if (nbytes < free_global_mem) {
        break;
      }
      mem_pool_sz--;
    }

    if (mem_pool_sz <= 1) {
      size_t minimal_required_nbytes = ptr_max_sz * sizeof(DType);
      if (minimal_required_nbytes > free_global_mem) {
        // We absolutely run out of memory
        auto error_msg = std::string("no enough GPU memory: requested ") +
            std::to_string(minimal_required_nbytes) + ", available: " +
            std::to_string(free_global_mem) + ", ptr_max_sz: " +
            std::to_string(ptr_max_sz) + ", at " + __FILE__;
        throw std::runtime_error(error_msg);
      } else {
        // Let's try to allocate a single blob that is large enough to hold
        // all input tensors. Note that this is still an approximation, because
        // we may still hit hipErrorOutOfMemory error while allocating
        // memory for the output. We will rely on hipMalloc to throw out
        // an exception in such a case.
        shared_input_tensor = true;
        AllocateGaussianTensor(ptr_max_sz);
      }
      return 1;
    }
    return mem_pool_sz;
  }

  DType* AllocateGaussianTensor(int64_t size) {
    size_t length = size * sizeof(DType);
    blobs.emplace_back(length);
    DType* ptr = reinterpret_cast<DType*>(blobs.back().get());

    uint64_t seed = uniform_dist(gen);
    double mean = 0.f;
    double std = 1.f;

    cutlass::reference::device::BlockFillRandomGaussian(ptr, size, seed, mean,
                                                        std);

    return ptr;
  }

  int AllocateTensor(int64_t size, int64_t copy, bool is_output = false) {
    offsets.push_back(0);
    strides.push_back(size);
    copies.push_back(copy);
    DType *ptr;
    if (!is_output && shared_input_tensor) {
      ptr = reinterpret_cast<DType*>(blobs.back().get());
    } else {
      ptr = AllocateGaussianTensor(size * copy);
    }
    ptrs.push_back(reinterpret_cast<void*>(ptr));
    return ptrs.size() - 1;
  }

  DType* RequestTensorByIdx(int idx) {
    auto copy = copies.at(idx);
    auto offset = offsets.at(idx);
    auto stride = strides.at(idx);
    DType* ptr = reinterpret_cast<DType*>(ptrs.at(idx));
    ptr += offset;
    offset += stride;
    if (offset == copy * stride) {
        offset = 0;
    }
    offsets[idx] = offset;
    return ptr;
  }

  std::vector<int64_t> offsets;
  std::vector<int64_t> strides;
  std::vector<int64_t> copies;
  std::vector<void*> ptrs;
  std::vector<cutlass::DeviceAllocation<uint8_t> > blobs;
  std::mt19937 gen;
  std::uniform_int_distribution<int64_t> uniform_dist;
  // make a shared blob to hold all inputs in cases we do not have
  // enough GPU memory
  bool shared_input_tensor;
};


int main(int argc, char** argv) {
  int device_idx;
  hipDeviceProp_t device_properties;
  hipError_t result = hipGetDevice(&device_idx);
  auto memory_pool = std::make_unique<ProfilerMemoryPool<half>>();
  if (result != hipSuccess) {
    std::ostringstream errorStream;
    errorStream << "hipGetDevice() call failed! "
                << "Error code: " << hipGetErrorName(result)
                << " Error message: " << hipGetErrorString(result);
    throw std::runtime_error(errorStream.str());
  }

  result = hipGetDeviceProperties(&device_properties, device_idx);

  if (result != hipSuccess) {
    std::ostringstream errorStream;
    errorStream << "hipGetDeviceProperties() call failed! "
                << "Error code: " << hipGetErrorName(result)
                << " Error message: " << hipGetErrorString(result);
    throw std::runtime_error(errorStream.str());
  }

  
  int64_t B = std::atoi(argv[1]);
  int64_t M = std::atoi(argv[2]);
  int64_t N = std::atoi(argv[3]);
  int64_t K = std::atoi(argv[4]);


  int64_t a_dim0 = B;

  int64_t a_dim1 = M;

  int64_t a_dim2 = K;


  int64_t b_dim0 = B;

  int64_t b_dim1 = N;

  int64_t b_dim2 = K;


  int64_t c_dim0 = B;

  int64_t c_dim1 = M;

  int64_t c_dim2 = N;


  uint8_t* global_workspace_ = nullptr;
  hipStream_t stream = nullptr;

  
  // cast to int64_t to avoid overflow
  int64_t a_ptr_sz = 1;
  
    a_ptr_sz *= static_cast<int64_t>(a_dim0);
  
    a_ptr_sz *= static_cast<int64_t>(a_dim1);
  
    a_ptr_sz *= static_cast<int64_t>(a_dim2);
  

  int64_t b_ptr_sz = 1;
  
    b_ptr_sz *= static_cast<int64_t>(b_dim0);
  
    b_ptr_sz *= static_cast<int64_t>(b_dim1);
  
    b_ptr_sz *= static_cast<int64_t>(b_dim2);
  

  int64_t c_ptr_sz = 1;
  
    c_ptr_sz *= static_cast<int64_t>(c_dim0);
  
    c_ptr_sz *= static_cast<int64_t>(c_dim1);
  
    c_ptr_sz *= static_cast<int64_t>(c_dim2);
  

  // The value 1 is used to force ptr_max_sz to be non-zero
  int64_t ptr_max_sz = std::max<int64_t>({1, a_ptr_sz, b_ptr_sz, c_ptr_sz});
  size_t one_copy_sz = a_ptr_sz + b_ptr_sz + c_ptr_sz;


  int64_t mem_pool_sz = memory_pool->ComputeMemPoolSize(one_copy_sz, ptr_max_sz, device_properties.l2CacheSize);

  memory_pool->AllocateTensor(a_ptr_sz, mem_pool_sz);  // a_ptr: index 0
  memory_pool->AllocateTensor(b_ptr_sz, mem_pool_sz);  // b_ptr: index 1
  memory_pool->AllocateTensor(c_ptr_sz, mem_pool_sz, /*is_output*/true);  // c_ptr: index 2



  
  {
  
  GemmInstance_0 gemm_op_0;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_256x128_32x3_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_bmm(
      gemm_op_0,
      gemm_op_name,
      memory_pool.get(),
      global_workspace_,


      &a_dim0,

      &a_dim1,

      &a_dim2,


      &b_dim0,

      &b_dim1,

      &b_dim2,


      &c_dim0,

      &c_dim1,

      &c_dim2,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_1 gemm_op_1;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_128x256_32x3_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_bmm(
      gemm_op_1,
      gemm_op_name,
      memory_pool.get(),
      global_workspace_,


      &a_dim0,

      &a_dim1,

      &a_dim2,


      &b_dim0,

      &b_dim1,

      &b_dim2,


      &c_dim0,

      &c_dim1,

      &c_dim2,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_2 gemm_op_2;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_256x64_32x3_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_bmm(
      gemm_op_2,
      gemm_op_name,
      memory_pool.get(),
      global_workspace_,


      &a_dim0,

      &a_dim1,

      &a_dim2,


      &b_dim0,

      &b_dim1,

      &b_dim2,


      &c_dim0,

      &c_dim1,

      &c_dim2,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_3 gemm_op_3;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_256x64_32x4_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_bmm(
      gemm_op_3,
      gemm_op_name,
      memory_pool.get(),
      global_workspace_,


      &a_dim0,

      &a_dim1,

      &a_dim2,


      &b_dim0,

      &b_dim1,

      &b_dim2,


      &c_dim0,

      &c_dim1,

      &c_dim2,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_4 gemm_op_4;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_64x256_32x4_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_bmm(
      gemm_op_4,
      gemm_op_name,
      memory_pool.get(),
      global_workspace_,


      &a_dim0,

      &a_dim1,

      &a_dim2,


      &b_dim0,

      &b_dim1,

      &b_dim2,


      &c_dim0,

      &c_dim1,

      &c_dim2,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_5 gemm_op_5;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_128x128_32x3_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_bmm(
      gemm_op_5,
      gemm_op_name,
      memory_pool.get(),
      global_workspace_,


      &a_dim0,

      &a_dim1,

      &a_dim2,


      &b_dim0,

      &b_dim1,

      &b_dim2,


      &c_dim0,

      &c_dim1,

      &c_dim2,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_6 gemm_op_6;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_128x128_32x4_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_bmm(
      gemm_op_6,
      gemm_op_name,
      memory_pool.get(),
      global_workspace_,


      &a_dim0,

      &a_dim1,

      &a_dim2,


      &b_dim0,

      &b_dim1,

      &b_dim2,


      &c_dim0,

      &c_dim1,

      &c_dim2,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_7 gemm_op_7;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_128x128_32x5_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_bmm(
      gemm_op_7,
      gemm_op_name,
      memory_pool.get(),
      global_workspace_,


      &a_dim0,

      &a_dim1,

      &a_dim2,


      &b_dim0,

      &b_dim1,

      &b_dim2,


      &c_dim0,

      &c_dim1,

      &c_dim2,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_8 gemm_op_8;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_128x64_32x6_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_bmm(
      gemm_op_8,
      gemm_op_name,
      memory_pool.get(),
      global_workspace_,


      &a_dim0,

      &a_dim1,

      &a_dim2,


      &b_dim0,

      &b_dim1,

      &b_dim2,


      &c_dim0,

      &c_dim1,

      &c_dim2,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_9 gemm_op_9;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_64x128_32x6_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_bmm(
      gemm_op_9,
      gemm_op_name,
      memory_pool.get(),
      global_workspace_,


      &a_dim0,

      &a_dim1,

      &a_dim2,


      &b_dim0,

      &b_dim1,

      &b_dim2,


      &c_dim0,

      &c_dim1,

      &c_dim2,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_10 gemm_op_10;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_64x64_32x10_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_bmm(
      gemm_op_10,
      gemm_op_name,
      memory_pool.get(),
      global_workspace_,


      &a_dim0,

      &a_dim1,

      &a_dim2,


      &b_dim0,

      &b_dim1,

      &b_dim2,


      &c_dim0,

      &c_dim1,

      &c_dim2,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_11 gemm_op_11;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_256x128_64x3_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_bmm(
      gemm_op_11,
      gemm_op_name,
      memory_pool.get(),
      global_workspace_,


      &a_dim0,

      &a_dim1,

      &a_dim2,


      &b_dim0,

      &b_dim1,

      &b_dim2,


      &c_dim0,

      &c_dim1,

      &c_dim2,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_12 gemm_op_12;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_128x256_64x3_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_bmm(
      gemm_op_12,
      gemm_op_name,
      memory_pool.get(),
      global_workspace_,


      &a_dim0,

      &a_dim1,

      &a_dim2,


      &b_dim0,

      &b_dim1,

      &b_dim2,


      &c_dim0,

      &c_dim1,

      &c_dim2,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_13 gemm_op_13;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_256x64_64x4_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_bmm(
      gemm_op_13,
      gemm_op_name,
      memory_pool.get(),
      global_workspace_,


      &a_dim0,

      &a_dim1,

      &a_dim2,


      &b_dim0,

      &b_dim1,

      &b_dim2,


      &c_dim0,

      &c_dim1,

      &c_dim2,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_14 gemm_op_14;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_64x256_64x4_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_bmm(
      gemm_op_14,
      gemm_op_name,
      memory_pool.get(),
      global_workspace_,


      &a_dim0,

      &a_dim1,

      &a_dim2,


      &b_dim0,

      &b_dim1,

      &b_dim2,


      &c_dim0,

      &c_dim1,

      &c_dim2,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_15 gemm_op_15;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_128x128_64x4_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_bmm(
      gemm_op_15,
      gemm_op_name,
      memory_pool.get(),
      global_workspace_,


      &a_dim0,

      &a_dim1,

      &a_dim2,


      &b_dim0,

      &b_dim1,

      &b_dim2,


      &c_dim0,

      &c_dim1,

      &c_dim2,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_16 gemm_op_16;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_256x64_64x3_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_bmm(
      gemm_op_16,
      gemm_op_name,
      memory_pool.get(),
      global_workspace_,


      &a_dim0,

      &a_dim1,

      &a_dim2,


      &b_dim0,

      &b_dim1,

      &b_dim2,


      &c_dim0,

      &c_dim1,

      &c_dim2,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_17 gemm_op_17;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_64x256_64x3_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_bmm(
      gemm_op_17,
      gemm_op_name,
      memory_pool.get(),
      global_workspace_,


      &a_dim0,

      &a_dim1,

      &a_dim2,


      &b_dim0,

      &b_dim1,

      &b_dim2,


      &c_dim0,

      &c_dim1,

      &c_dim2,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_18 gemm_op_18;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_128x128_64x3_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_bmm(
      gemm_op_18,
      gemm_op_name,
      memory_pool.get(),
      global_workspace_,


      &a_dim0,

      &a_dim1,

      &a_dim2,


      &b_dim0,

      &b_dim1,

      &b_dim2,


      &c_dim0,

      &c_dim1,

      &c_dim2,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_19 gemm_op_19;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_128x64_64x3_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_bmm(
      gemm_op_19,
      gemm_op_name,
      memory_pool.get(),
      global_workspace_,


      &a_dim0,

      &a_dim1,

      &a_dim2,


      &b_dim0,

      &b_dim1,

      &b_dim2,


      &c_dim0,

      &c_dim1,

      &c_dim2,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_20 gemm_op_20;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_64x128_64x3_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_bmm(
      gemm_op_20,
      gemm_op_name,
      memory_pool.get(),
      global_workspace_,


      &a_dim0,

      &a_dim1,

      &a_dim2,


      &b_dim0,

      &b_dim1,

      &b_dim2,


      &c_dim0,

      &c_dim1,

      &c_dim2,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }

  {
  
  GemmInstance_21 gemm_op_21;
  const char *gemm_op_name = "cutlass_tensorop_f16_s16816gemm_f16_64x64_64x5_tn_align_8_8";
  int ret = 0;
  try {
  ret = benchmark_bmm(
      gemm_op_21,
      gemm_op_name,
      memory_pool.get(),
      global_workspace_,


      &a_dim0,

      &a_dim1,

      &a_dim2,


      &b_dim0,

      &b_dim1,

      &b_dim2,


      &c_dim0,

      &c_dim1,

      &c_dim2,

      stream
  );
  } catch (...) {}
  if (ret != 0)
    return ret;
  
  }
  return 0;
}