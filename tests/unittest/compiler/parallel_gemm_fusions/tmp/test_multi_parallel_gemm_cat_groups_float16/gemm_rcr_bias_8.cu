#include "hip/hip_runtime.h"

#include <iostream>
#include <memory>
#include <random>
#include <vector>
#include <iostream>
#include <hip/hip_bf16.h>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm_universal.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/epilogue/thread/linear_combination_silu.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "cutlass/util/device_memory.h"

#include "cutlass/gemm/gemm.h"
#include "cutlass/numeric_types.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/epilogue/collective/collective_builder.hpp"

using bfloat16 = hip_bfloat16;


using elem_input_type = cutlass::half_t;
using elem_output_type = cutlass::half_t;

#define CUTLASS_CHECK(status)                                                         \
  {                                                                                   \
    cutlass::Status error = status;                                                   \
    if (error != cutlass::Status::kSuccess) {                                         \
      auto msg = std::string("[") + __FILE__ + "] Got cutlass error: " +              \
          cutlassGetStatusString(error) + " at: " + std::to_string(__LINE__);         \
      std::cerr << msg << std::endl;                                                  \
      throw std::runtime_error(msg);                                                  \
    }                                                                                 \
  }



  // Gemm operator cutlass_tensorop_f16_s16816gemm_f16_64x64_64x5_tn_align8
  using Operation_cutlass_tensorop_f16_s16816gemm_f16_64x64_64x5_tn_align8 = cutlass::gemm::device::GemmUniversal<
    cutlass::half_t, cutlass::layout::RowMajor,
    cutlass::half_t, cutlass::layout::ColumnMajor,
    cutlass::half_t, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 64>,
    cutlass::gemm::GemmShape<32, 32, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,
      8,
      float,
      float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    5,
    8,
    8,
    
    cutlass::arch::OpMultiplyAdd
    
  >;

using f4ebbe7825166ed2ddedb08c7b746bf5ab8d4e970 = Operation_cutlass_tensorop_f16_s16816gemm_f16_64x64_64x5_tn_align8;

void gemm_rcr_bias_8 (
    void* a_ptr,
    void* b_ptr,
    void* bias_ptr,
    void* c_ptr,
    uint8_t* workspace,
    int split_k,
    int64_t* a_dim0,
    int64_t* a_dim1,
    int64_t* b_dim0,
    int64_t* b_dim1,
    int64_t* c_dim0,
    int64_t* c_dim1,
  hipStream_t stream
  ) {
  
 int64_t M = (*a_dim0);

 int64_t N = (*b_dim0);

 int64_t K = (*a_dim1);
  
  int64_t input_a_batch_stride = M * K;
  int64_t input_a_stride = K;
  int64_t input_a_offset = 0; // default to 0
  int64_t input_b_batch_stride = N * K;
  int64_t input_b_stride = K;
  int64_t input_b_offset = 0; // default to 0
    
  
  
  int64_t output_stride = 768;
  int64_t output_offset = 256;
  
    
  
  
  int64_t a_size = 1;

    a_size *= *a_dim0;

    a_size *= *a_dim1;

  if (a_size != 0 && !a_ptr) {
    throw std::runtime_error("input a is null!");
  }

  int64_t b_size = 1;

    b_size *= *b_dim0;

    b_size *= *b_dim1;

  if (b_size != 0 && !b_ptr) {
    throw std::runtime_error("input b is null!");
  }

  int64_t c_size = 1;

    c_size *= *c_dim0;

    c_size *= *c_dim1;

  if (c_size != 0) {
    if (!c_ptr) {
      throw std::runtime_error("input c is null!");
    }
  } else {
    // output is empty and safe to return
    return;
  }

  // One of the input tensor are empty
  if (a_size == 0 || b_size == 0) {
    return;
  }

  if (!bias_ptr) {
    throw std::runtime_error("bias_ptr is null!");
  }

  
  if (M == 256 && N == 128 && K == 120) {
    
//  TODO: cast to right dtype
    using ElementComputeEpilogue = typename f4ebbe7825166ed2ddedb08c7b746bf5ab8d4e970::ElementAccumulator;

    using coord_t = cutlass::gemm::GemmCoord::Index;
    typename f4ebbe7825166ed2ddedb08c7b746bf5ab8d4e970::Arguments arguments;

    if constexpr (cutlass::gemm::detail::IsCutlass3GemmKernel<typename f4ebbe7825166ed2ddedb08c7b746bf5ab8d4e970::GemmKernel>::value) {
    arguments = {

    };
    } else {
    arguments = {

    cutlass::gemm::GemmUniversalMode::kGemm,                 // GemmUniversalMode mode
    cutlass::gemm::GemmCoord{
        static_cast<coord_t>(M),
        static_cast<coord_t>(N),
        static_cast<coord_t>(K)
    },                                                       // GemmCoord problem_size
    split_k,                                                 // int batch_count
    {ElementComputeEpilogue(1), ElementComputeEpilogue(1)},  // typename EpilogueOutputOp::Params epilogue
    (cutlass::half_t*)(a_ptr) + input_a_offset,          // void const * ptr_A
    (cutlass::half_t*)(b_ptr) + input_b_offset,          // void const * ptr_B
    (cutlass::half_t*)(bias_ptr),                        // void const * ptr_C
    (cutlass::half_t*)(c_ptr) + output_offset,          // void * ptr_D
    input_a_batch_stride,                                    // int64_t batch_stride_A
    input_b_batch_stride,                                    // int64_t batch_stride_B
    N,                                                       // int64_t batch_stride_C
    M * N,                                                   // int64_t batch_stride_D
    input_a_stride,                                          // typename LayoutA::Stride::LongIndex lda
    input_b_stride,                                          // typename LayoutB::Stride::LongIndex ldb
    0,                                                       // typename LayoutC::Stride::LongIndex ldc
    output_stride,                                           // typename LayoutC::Stride::LongIndex ldd
    };
    }


    f4ebbe7825166ed2ddedb08c7b746bf5ab8d4e970 gemm_op;

    auto status = gemm_op.can_implement(arguments);
    CUTLASS_CHECK(status);
    status = gemm_op.initialize(arguments, workspace, stream);
    CUTLASS_CHECK(status);
    status = gemm_op(stream);
    CUTLASS_CHECK(status);
    return;
  }
      std::cout << "input_ndims0: " << *a_dim0 << std::endl;
      std::cout << "input_ndims1: " << *a_dim1 << std::endl;
      std::cout << "weight_ndims0: " << *b_dim0 << std::endl;
      std::cout << "weight_ndims1: " << *b_dim1 << std::endl;
      std::cout << "output_ndims0: " << *c_dim0 << std::endl;
      std::cout << "output_ndims1: " << *c_dim1 << std::endl;
  throw std::runtime_error(
      "Unsupported workload for this gemm_rcr_bias_8 specialization."
  );
}